#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hipDNN.h>

// Check CUDA and cuDNN calls for errors
#define CHECK_CUDA(call) {\
    hipError_t err = call;\
    if (err != hipSuccess) {\
        std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << " - " << hipGetErrorString(err) << std::endl;\
        exit(EXIT_FAILURE);\
    }\
}

#define CHECK_CUDNN(call) {\
    hipdnnStatus_t err = call;\
    if (err != HIPDNN_STATUS_SUCCESS) {\
        std::cerr << "cuDNN error at " << __FILE__ << ":" << __LINE__ << " - " << hipdnnGetErrorString(err) << std::endl;\
        exit(EXIT_FAILURE);\
    }\
}

// Function to print the first 2x2 of a tensor
void printMatrixPreview(const float* matrix, int rows, int cols, const std::string& name) {
    std::cout << name << " (First 2*2):\n";
    for (int i = 0; i < std::min(rows, 2); ++i) {
        for (int j = 0; j < std::min(cols, 2); ++j) {
            std::cout << matrix[i * cols + j] << " ";
        }
        if (cols > 2)
            std::cout << "...";
        std::cout << "\n";
    }
    if (rows > 2)
        std::cout << "...\n";
}

int main() {
    // Input dimensions: N (batch size), C (channels), H (height), W (width)
    int N = 1, C = 1, H = 5, W = 5;
    int K = 1, R = 3, S = 3; // Output channels, kernel height, kernel width
    int pad_h = 1, pad_w = 1, stride_h = 1, stride_w = 1;

    // Allocate and initialize host memory for input and kernel
    std::vector<float> h_input(N * C * H * W);
    std::vector<float> h_kernel(K * C * R * S);
    std::vector<float> h_output;

    // Initialize input and kernel with some values
    for (int i = 0; i < N * C * H * W; ++i) {
        h_input[i] = static_cast<float>(i + 1); // Example: Sequential values
    }
    for (int i = 0; i < K * C * R * S; ++i) {
        h_kernel[i] = 0.5f; // Example: All elements are 0.5
    }

    // cuDNN handles
    hipdnnHandle_t cudnn;
    hipdnnTensorDescriptor_t inputDesc, outputDesc;
    hipdnnFilterDescriptor_t filterDesc;
    hipdnnConvolutionDescriptor_t convDesc;

    CHECK_CUDNN(hipdnnCreate(&cudnn));
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&inputDesc));
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&outputDesc));
    CHECK_CUDNN(hipdnnCreateFilterDescriptor(&filterDesc));
    CHECK_CUDNN(hipdnnCreateConvolutionDescriptor(&convDesc));

    // Input tensor descriptor
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(inputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N, C, H, W));

    // Kernel descriptor
    CHECK_CUDNN(hipdnnSetFilter4dDescriptor(filterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, K, C, R, S));

    // Convolution descriptor
    CHECK_CUDNN(hipdnnSetConvolution2dDescriptor(convDesc, pad_h, pad_w, stride_h, stride_w, 1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

    // Output dimensions
    int outN, outC, outH, outW;
    CHECK_CUDNN(hipdnnGetConvolution2dForwardOutputDim(convDesc, inputDesc, filterDesc, &outN, &outC, &outH, &outW));

    h_output.resize(outN * outC * outH * outW);

    // Output tensor descriptor
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(outputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, outN, outC, outH, outW));

    // Allocate device memory
    float *d_input, *d_kernel, *d_output;
    CHECK_CUDA(hipMalloc(&d_input, N * C * H * W * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_kernel, K * C * R * S * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_output, outN * outC * outH * outW * sizeof(float)));

    // Copy data to device
    CHECK_CUDA(hipMemcpy(d_input, h_input.data(), N * C * H * W * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_kernel, h_kernel.data(), K * C * R * S * sizeof(float), hipMemcpyHostToDevice));

    // Convolution algorithm
    hipdnnConvolutionFwdAlgo_t algo;
    CHECK_CUDNN(hipdnnGetConvolutionForwardAlgorithm(cudnn, inputDesc, filterDesc, convDesc, outputDesc, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));

    // Workspace allocation
    size_t workspaceSize = 0;
    CHECK_CUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn, inputDesc, filterDesc, convDesc, outputDesc, algo, &workspaceSize));
    void* d_workspace = nullptr;
    if (workspaceSize > 0) {
        CHECK_CUDA(hipMalloc(&d_workspace, workspaceSize));
    }

    // Alpha and beta values
    const float alpha = 1.0f, beta = 0.0f;

    // Record start time
    auto start = std::chrono::high_resolution_clock::now();

    // Perform convolution
    CHECK_CUDNN(hipdnnConvolutionForward(cudnn, &alpha, inputDesc, d_input, filterDesc, d_kernel, convDesc, algo, d_workspace, workspaceSize, &beta, outputDesc, d_output));

    // Record end time
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float, std::milli> elapsed = end - start;

    // Copy result back to host
    CHECK_CUDA(hipMemcpy(h_output.data(), d_output, outN * outC * outH * outW * sizeof(float), hipMemcpyDeviceToHost));

    // Print matrices
    printMatrixPreview(h_input.data(), H, W, "Input");
    std::cout << "\n";
    printMatrixPreview(h_kernel.data(), R, S, "Kernel");
    std::cout << "\n";
    printMatrixPreview(h_output.data(), outH, outW, "Output");

    std::cout << "\n";
    std::cout << "Convolution time: " << elapsed.count() << " ms\n";

    // Cleanup
    hipFree(d_input);
    hipFree(d_kernel);
    hipFree(d_output);
    if (workspaceSize > 0) hipFree(d_workspace);
    hipdnnDestroyTensorDescriptor(inputDesc);
    hipdnnDestroyTensorDescriptor(outputDesc);
    hipdnnDestroyFilterDescriptor(filterDesc);
    hipdnnDestroyConvolutionDescriptor(convDesc);
    hipdnnDestroy(cudnn);

    return 0;
}
